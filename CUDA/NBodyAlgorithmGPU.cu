#include "hip/hip_runtime.h"
#include "NBodyAlgorithmGPU.cuh"

// szop�shegy van az eps2 �tad�s�val (nem akarom argumentumk�nt �tadni minden egyes alkalommal


void NBodyAlgorithmGPU::unpackBodies(std::vector<Body> &bodies) {
#pragma unroll
    for (int i = 0; i < mp_properties->numBody; i++) {
        mph_mass[i] = bodies.at(i).mass;
        mph_position[i] = bodies.at(i).position;
        mph_velocity[i] = bodies.at(i).velocity;
        mph_acceleration[i] = bodies.at(i).acceleration;
    }
}

void NBodyAlgorithmGPU::packBodies(std::vector<Body> &bodies) {
#pragma unroll
    for (int i = 0; i < mp_properties->numBody; i++) {
        //bodies.at(i).mass = mph_mass[i];
        bodies.at(i).position = mph_position[i];
        bodies.at(i).velocity = mph_velocity[i];
        bodies.at(i).acceleration = mph_acceleration[i];
    }
}

void NBodyAlgorithmGPU::init(std::vector<Body> &bodies) {
    // Van-e CUDA kompatibilis GPU?
    int numDevice;
    checkCudaError(hipGetDeviceCount(&numDevice));
    if (!numDevice) {
        std::cerr << "No CUDA compatible device detected. Aborting..." << std::endl;
        exit(0);
    }

    for (int i = 0; i < numDevice; i++) {
        hipDeviceProp_t prop;
        checkCudaError(hipGetDeviceProperties(&prop, i));
        std::cout << "Device Number: " << i << std::endl;
        std::cout << "   Device name: " << prop.name << std::endl;
        std::cout << "   Device Clock Rate (MHz): " << prop.clockRate / 1e3 << std::endl;
        std::cout << "   Multiprocessor Count: " << prop.multiProcessorCount << std::endl;
        std::cout << std::endl;
        std::cout << "   Total Global Memory (MB): " << prop.totalGlobalMem / 1024 / 1024 << std::endl;
        std::cout << "   Total Constant Memory (kB): " << prop.totalConstMem / 1024 << std::endl;
        std::cout << "   Memory Clock Rate (MHz): " << prop.memoryClockRate / 1e3 << std::endl;
        std::cout << "   Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
        std::cout << "   Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6 << std::endl;
        std::cout << std::endl;
        std::cout << "   Max Grid Size: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << " )" << std::endl;
        std::cout << "   Max Threads Dimension: (" << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << " )" << std::endl;
        std::cout << "   Max Threads Per Block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "   Max Threads Per Multiprocessor: " << prop.maxThreadsPerMultiProcessor << std::endl;
        std::cout << std::endl;
        std::cout << "   Registers Per Block: " << prop.regsPerBlock << std::endl;
        std::cout << "   Shared Memory per Block: " << prop.sharedMemPerBlock << std::endl;
    }
    
    // GPU kofigur�ci�
    checkCudaError(hipSetDevice(0));
    //checkCudaError(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&matMulNaive), hipFuncCachePreferL1));
    //checkCudaError(hipFuncSetCacheConfig(reinterpret_cast<const void*>(&matMulNaive), hipFuncCachePreferShared));

    // Testek kicsomagol�sa t�mb�kbe, a host mem�ri�ba
    unpackBodies(bodies);

    // Mem�riaallok�ci�
    checkCudaError(hipMalloc((void**)&mpd_mass, mp_properties->numBody * sizeof(float)));
    checkCudaError(hipMalloc((void**)&mpd_position, mp_properties->numBody * sizeof(float3)));
    checkCudaError(hipMalloc((void**)&mpd_acceleration, mp_properties->numBody * sizeof(float3)));
    if (mp_properties->mode == GUI) {
        checkCudaError(hipMalloc((void**)&mpd_numNeighbours, mp_properties->numBody * sizeof(float)));
    }
    
    // M�sol�s GPU global mem�ri�ba
    checkCudaError(hipMemcpy(mpd_mass, mph_mass, mp_properties->numBody * sizeof(float), hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(mpd_position, mph_position, mp_properties->numBody * sizeof(float3), hipMemcpyHostToDevice));

    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(d_numBody), &(mp_properties->numBody), sizeof(mp_properties->numBody)));

    setKernelParameters();
}

void NBodyAlgorithmGPU::destroy() {
    // Allok�lt mem�ria felszabad�t�sa
    checkCudaError(hipFree(mpd_mass));
    checkCudaError(hipFree(mpd_position));
    checkCudaError(hipFree(mpd_acceleration));

    checkCudaError(hipDeviceReset());
}

void NBodyAlgorithmGPU::setKernelParameters() {
    int numBody = mp_properties->numBody;
    float minOccupancy = 75.0f; // %

    hipDeviceProp_t prop;
    checkCudaError(hipGetDeviceProperties(&prop, 0));

    int numMultiProcessor = prop.multiProcessorCount;
    int maxResidentThreadBlockPerMultiProcessor;
    switch (prop.major) // Compute capability
    {
    case 2: // 2.x
        maxResidentThreadBlockPerMultiProcessor = 8;
        break;
    case 3:
        maxResidentThreadBlockPerMultiProcessor = 16;
        break;
    case 5:
        maxResidentThreadBlockPerMultiProcessor = 32;
        break;
    case 6:
        maxResidentThreadBlockPerMultiProcessor = 32;
        break;
    default:
        maxResidentThreadBlockPerMultiProcessor = 8;
        break;
    }
    
    int maxActiveThreadBlocks = maxResidentThreadBlockPerMultiProcessor * numMultiProcessor;    // 16
    int maxThreadsPerMultiProcessor = prop.maxThreadsPerMultiProcessor;
    int maxActiveThreads = maxThreadsPerMultiProcessor * numMultiProcessor;     // 3072
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;       // 1024
    int wrapSize = prop.warpSize;   // 32

    uint3 tempDim = { prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2] };
    dim3 maxGridDim = tempDim;
    tempDim = { prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] };
    dim3 maxThreadDim = tempDim;

    int maxSharedMemoryPerBlock = prop.sharedMemPerBlock;   // �sszesen 48k
    int maxRegistersPerBlock = prop.regsPerBlock;           // �sszesen 32768
    
    // Compute Capability:      2.1
    // Max register per block:  32768
    // Max register per thread: 63
    
    int optimalThreadsPerBlock = maxActiveThreads / minOccupancy * 100 / maxActiveThreadBlocks; // 256
    // 16x16-os kernel ind�t�s�nak nincs nagyon �rtelme ezzel a tile-os, kommunik�ci� n�lk�li m�dszerrel
    // K�tdimenzi�s kioszt�ssal a Z �rt�k�t fixen 1-re �ll�tom
    unsigned int threadBlockX = wrapSize;    // 32
    unsigned int threadBlockY = optimalThreadsPerBlock / wrapSize; // 8
    unsigned int threadBlockZ = 1;

    unsigned int blockGridX = numBody / threadBlockX + ((numBody % threadBlockX) != 0);
    unsigned int blockGridY = numBody / threadBlockY + ((numBody % threadBlockY) != 0);
    unsigned int blockGridZ = 1;
    if (threadBlockX > maxThreadDim.x || threadBlockX > maxThreadDim.y || threadBlockX > maxThreadDim.z) {
        std::cout << "Thread blocks contain more threads than the max value." << std::endl;
        exit(0);
    }
    if (blockGridX > maxGridDim.x || blockGridY > maxGridDim.y || blockGridZ > maxGridDim.z) {
        std::cout << "Grid contain more thread blocks than the max value." << std::endl;
        exit(0);
    }
    m_gridSize = { blockGridX, blockGridY, blockGridZ };
    m_threadBlockSize = { threadBlockX, threadBlockY, threadBlockZ };
    m_sharedMemorySize = optimalThreadsPerBlock * (3 + 1) * sizeof(float);    // 3 pos, 1 mass
}