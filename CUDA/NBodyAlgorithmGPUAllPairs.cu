#include "hip/hip_runtime.h"
#include "NBodyAlgorithmGPUAllPairs.cuh"

// Kellene egy k�l�n f�ggv�ny valahova magasabb szinten ami ellen�rzi, hogy van-e alkalmas GPU
// Kellene egy�b utility f�ggv�ny a device param�terek ki�rat�s�ra is, de ha lehet ne szemetelj�k tele a k�dot CUDA k�ddal
void NBodyAlgorithmGPUAllPairs::advance(std::vector<Body> &bodies) {

    advanceKernel <<< m_gridSize, m_threadBlockSize, m_sharedMemorySize >>> (mpd_position, mpd_mass, mpd_acceleration);

    hipError_t kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(kernelStatus) << std::endl;
    }

    checkCudaError(hipDeviceSynchronize());


    // TODO: visszaolvasni a kernel �ltal kisz�molt gyorsul�s�rt�keket
    // Update-elni a sebess�get �s a poz�ci�t
}

#define EPS2 10.0f

// Valami�rt nem tetszik az NVCC-nek ha m�sik f�jlban van deklar�lva �s defini�lva a kernel �ltal h�vogatott __device__ f�ggv�ny
__device__ float3 calculateAcceleration(const float3 posI, const float massJ, const float3 posJ, float3 accSumI) {
    float3 r;

    r.x = posJ.x - posI.x;
    r.y = posJ.y - posI.y;
    r.z = posJ.z - posI.z;

    float rabs = sqrt(r.x * r.x + r.y * r.y + r.z * r.z + EPS2);
    float rabsInv = 1.0f / (rabs * rabs * rabs);
    float temp = massJ * rabsInv;

    // A t�megbe bele van olvasztva a G
    // Az rabsInv-be beleraktum a massJ-t, hogy ne kelljen mindig �sszeszorozni
    // Be kelle hozni az gyorul�s �rt�kek akkumul�l�s�t, mert float3/float4-gyel val� m�veleteket nem t�mogatja a CUDA C
    // Szint�n nem elhanyagolhat� hogy MAC m�veletet ki kell haszn�lni, nem aj�nlott az akkumul�l�st k�l�n elv�gezni
    accSumI.x += r.x * temp;
    accSumI.y += r.y * temp;
    accSumI.z += r.z * temp;
    return accSumI;
}

__device__ float3 tileCalculateAcceleration(const float3 posI, float3 accI) {
    float3 accSumI = accI;
    extern __shared__ float sh_mass[];
    extern __shared__ float3 sh_pos[];
    for (int i = 0; i < blockDim.x; i++) {
        accSumI = calculateAcceleration(posI, sh_mass[i], sh_pos[i], accSumI);
    }
    return accSumI;
}


__global__ void advanceKernel(float3 *g_pos, float *g_mass, float3 *g_acc) {
    extern __shared__ float sh_mass[];
    extern __shared__ float3 sh_pos[];

    //float massI;
    float3 posI;
    float3 accI;

    int gridID = blockIdx.x * blockDim.x + threadIdx.x;
    //massI = g_mass[gridID];
    posI = g_pos[gridID];
    accI = { 0.0f, 0.0f, 0.0f };

    for (int i = 0, tile = 0; i < d_numBody; i += blockDim.y, tile++) {
        int tileID = tile * blockDim.y + threadIdx.x;
        // Ha esetleg nem n�gyzetes TB lenne, nem kell minden threadnek bet�ltenie a shared mem�ria
        if (threadIdx.x < blockDim.y) {
            sh_mass[threadIdx.x] = g_mass[tileID];
            sh_pos[threadIdx.x] = g_pos[tileID];
        }
        __syncthreads();    // shared mem�ria t�lt�se
        accI = tileCalculateAcceleration(posI, accI);
        __syncthreads();    // ne kezd�dj�n �jra a shared mem�ria felt�lt�se
    }
    g_acc[gridID] = accI;
}