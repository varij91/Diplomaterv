#include "hip/hip_runtime.h"
#include "NBodyAlgorithmGPUAllPairs.cuh"

void NBodyAlgorithmGPUAllPairs::updateBodies(std::vector<Body> &bodies) {
    float stepTime2 = mp_properties->stepTime * mp_properties->stepTime;
#pragma unroll
    for (int i = 0; i < mp_properties->numBody; i++) {
        mph_position[i].x += mph_velocity[i].x * mp_properties->stepTime + mph_acceleration[i].x * stepTime2;
        mph_position[i].y += mph_velocity[i].y * mp_properties->stepTime + mph_acceleration[i].y * stepTime2;
        mph_position[i].z += mph_velocity[i].z * mp_properties->stepTime + mph_acceleration[i].z * stepTime2;

        mph_velocity[i].x = mph_velocity[i].x * mp_properties->VELOCITY_DAMPENING + mph_acceleration[i].x * mp_properties->stepTime;
        mph_velocity[i].y = mph_velocity[i].y * mp_properties->VELOCITY_DAMPENING + mph_acceleration[i].y * mp_properties->stepTime;
        mph_velocity[i].z = mph_velocity[i].z * mp_properties->VELOCITY_DAMPENING + mph_acceleration[i].z * mp_properties->stepTime;
    }
    if (mp_properties->mode == GUI) {
        packBodies(bodies);
        std::vector<unsigned int> temp(mph_numNeighbours, mph_numNeighbours + mp_properties->numBody);
        mp_properties->numNeighbours = temp;
    }
}

// Kellene egy k�l�n f�ggv�ny valahova magasabb szinten ami ellen�rzi, hogy van-e alkalmas GPU
// Kellene egy�b utility f�ggv�ny a device param�terek ki�rat�s�ra is, de ha lehet ne szemetelj�k tele a k�dot CUDA k�ddal
void NBodyAlgorithmGPUAllPairs::advance(std::vector<Body> &bodies) {
    if (mp_properties->mode == GUI) {
        //advanceKernelWithColor << < m_gridSize, m_threadBlockSize, m_sharedMemorySize + 12 >> > (mpd_position, mpd_mass, mpd_acceleration, mpd_numNeighbours);
    }
    else {
        advanceKernel << < m_gridSize, m_threadBlockSize, m_sharedMemorySize >> > (mpd_position, mpd_mass, mpd_acceleration, mpd_numBodies, mpd_eps2);
    }

    hipError_t kernelStatus = hipGetLastError();
    if (kernelStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(kernelStatus) << std::endl;
    }

    //ITT FAGY KI
    checkCudaError(hipDeviceSynchronize());

    // TODO: visszaolvasni a kernel �ltal kisz�molt gyorsul�s�rt�keket
    // Update-elni a sebess�get �s a poz�ci�t

    checkCudaError(hipMemcpy(mph_acceleration, mpd_acceleration, mp_properties->numBody * sizeof(float3), hipMemcpyDeviceToHost));
    if (mp_properties->mode == GUI) {
        checkCudaError(hipMemcpy(mph_numNeighbours, mpd_numNeighbours, mp_properties->numBody * sizeof(float), hipMemcpyDeviceToHost));
    }
    updateBodies(bodies);
}

// Valami�rt nem tetszik az NVCC-nek ha m�sik f�jlban van deklar�lva �s defini�lva a kernel �ltal h�vogatott __device__ f�ggv�ny
__device__ float3 calculateAcceleration(const float3 posI, const float massJ, const float3 posJ, float3 accSumI, const int eps2) {
    float3 r;

    r.x = posJ.x - posI.x;
    r.y = posJ.y - posI.y;
    r.z = posJ.z - posI.z;

    float rabs = sqrt(r.x * r.x + r.y * r.y + r.z * r.z + eps2);
    float rabsInv = 1.0f / (rabs * rabs * rabs);
    float temp = massJ * rabsInv;

    // A t�megbe bele van olvasztva a G
    // Az rabsInv-be beleraktum a massJ-t, hogy ne kelljen mindig �sszeszorozni
    // Be kelle hozni az gyorul�s �rt�kek akkumul�l�s�t, mert float3/float4-gyel val� m�veleteket nem t�mogatja a CUDA C
    // Szint�n nem elhanyagolhat� hogy MAC m�veletet ki kell haszn�lni, nem aj�nlott az akkumul�l�st k�l�n elv�gezni
    accSumI.x += r.x * temp;
    accSumI.y += r.y * temp;
    accSumI.z += r.z * temp;
    return accSumI;
}

__device__ float3 tileCalculateAcceleration(const float3 posI, float3 accI, const int eps2) {
    float3 accSumI = accI;
    extern __shared__ float sh_mass[];
    extern __shared__ float3 sh_pos[];
    for (int i = 0; i < blockDim.x; i++) {
        accSumI = calculateAcceleration(posI, sh_mass[i], sh_pos[i], accSumI, eps2);
    }
    return accSumI;
}

__global__ void advanceKernel(float3 *g_pos, float *g_mass, float3 *g_acc, int *g_numBodies, float *g_eps2) {
    extern __shared__ float sh_mass[];
    extern __shared__ float3 sh_pos[];

    //float massI;
    float3 posI;
    float3 accI;
    int numBodies = (*g_numBodies);
    int eps2 = (*g_eps2);

    int gridID = blockIdx.x * blockDim.x + threadIdx.x;
    //massI = g_mass[gridID];
    posI = g_pos[gridID];
    accI = { 0.0f, 0.0f, 0.0f };
    printf("111111111\n");
    for (int i = 0, tile = 0; i < numBodies; i += blockDim.y, tile++) {
        int tileID = tile * blockDim.y + threadIdx.x;
        printf("222222222\n");
        // Ha esetleg nem n�gyzetes TB lenne, nem kell minden threadnek bet�ltenie a shared mem�ria
        if (threadIdx.x < blockDim.y) {
            sh_mass[threadIdx.x] = g_mass[tileID];
            sh_pos[threadIdx.x] = g_pos[tileID];
            //printf("(%f, %f, %f)", sh_pos[threadIdx.x].x, sh_pos[threadIdx.x].y, sh_pos[threadIdx.x].z);
           
        }
        __syncthreads();    // shared mem�ria t�lt�se
        accI = tileCalculateAcceleration(posI, accI, eps2);
        __syncthreads();    // ne kezd�dj�n �jra a shared mem�ria felt�lt�se
    }
    //printf("(%f, %f, %f)", accI.x, accI.y, accI.z);
    g_acc[gridID] = accI;
}


/*__device__ float3 calculateAccelerationWithColor(const float3 posI, const float massJ, const float3 posJ, float3 accSumI, float *numNeighbours) {
    float3 r;

    r.x = posJ.x - posI.x;
    r.y = posJ.y - posI.y;
    r.z = posJ.z - posI.z;

    float rabs = sqrt(r.x * r.x + r.y * r.y + r.z * r.z + d_EPS2);
    float rabsInv = 1.0f / (rabs * rabs * rabs);
    float temp = massJ * rabsInv;
    (*numNeighbours) = (rabs < d_POSITION_SCALE) ? (*numNeighbours) + 1 : (*numNeighbours);
    // A t�megbe bele van olvasztva a G
    // Az rabsInv-be beleraktum a massJ-t, hogy ne kelljen mindig �sszeszorozni
    // Be kelle hozni az gyorul�s �rt�kek akkumul�l�s�t, mert float3/float4-gyel val� m�veleteket nem t�mogatja a CUDA C
    // Szint�n nem elhanyagolhat� hogy MAC m�veletet ki kell haszn�lni, nem aj�nlott az akkumul�l�st k�l�n elv�gezni
    accSumI.x += r.x * temp;
    accSumI.y += r.y * temp;
    accSumI.z += r.z * temp;
    return accSumI;
}

__device__ float3 tileCalculateAccelerationWithColor(const float3 posI, float3 accI, float *numNeighbours) {
    float3 accSumI = accI;
    extern __shared__ float sh_mass[];
    extern __shared__ float3 sh_pos[];
    for (int i = 0; i < blockDim.x; i++) {
        accSumI = calculateAccelerationWithColor(posI, sh_mass[i], sh_pos[i], accSumI, numNeighbours);
    }
    return accSumI;
}

__global__ void advanceKernelWithColor(float3 *g_pos, float *g_mass, float3 *g_acc, float *g_numNeighbours) {
    extern __shared__ float sh_mass[];
    extern __shared__ float3 sh_pos[];

    //float massI;
    float3 posI;
    float3 accI;
    float numNeighbours = 0.0f;

    int gridID = blockIdx.x * blockDim.x + threadIdx.x;
    //massI = g_mass[gridID];
    posI = g_pos[gridID];
    accI = { 0.0f, 0.0f, 0.0f };

    for (int i = 0, tile = 0; i < d_NUM_BODY; i += blockDim.y, tile++) {
        int tileID = tile * blockDim.y + threadIdx.x;
        // Ha esetleg nem n�gyzetes TB lenne, nem kell minden threadnek bet�ltenie a shared mem�ria
        if (threadIdx.x < blockDim.y) {
            sh_mass[threadIdx.x] = g_mass[tileID];
            sh_pos[threadIdx.x] = g_pos[tileID];
        }
        __syncthreads();    // shared mem�ria t�lt�se
        accI = tileCalculateAccelerationWithColor(posI, accI, &numNeighbours);
        __syncthreads();    // ne kezd�dj�n �jra a shared mem�ria felt�lt�se
    }
    g_acc[gridID] = accI;
}*/
